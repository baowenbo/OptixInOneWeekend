
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>

// Kernel function to color the buffer according to the gradient
__global__ void color(float *buffer)
{
    int x = threadIdx.x;
    int y = blockIdx.x;
    int nx = blockDim.x;
    int ny = gridDim.x;
    float r = (float) x / nx;
    float g = (float) y / ny;
    float b = 0.2;
    float* triplet = buffer + 3*(blockIdx.x*blockDim.x + threadIdx.x);
    triplet[0] = r;
    triplet[1] = g;
    triplet[2] = b;
}

int main(void)
{
    int nx = 200;
    int ny = 100;
    float *buffer;

    // Allocate Unified Memory – accessible from CPU or GPU
    hipMallocManaged(&buffer, 3*nx*ny*sizeof(float));

    // Run kernel on nx*ny elements on the GPU
    color<<<ny, nx>>>(buffer);

    // Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();

    // Print the buffer out to command
    std::cout << "P3\n" << nx << " " << ny << "\n255\n";
    for (int j = ny-1; j >= 0; j--)
        for (int i = 0; i < nx; i++)
        {
            float* triplet = buffer + 3*(j*nx + i);
            int ir = int(255.99f * triplet[0]);
            int ig = int(255.99f * triplet[1]);
            int ib = int(255.99f * triplet[2]);
            std::cout << ir << " " << ig << " " << ib << "\n";
        }

    // Free memory
    hipFree(buffer);
  
    return 0;
}
