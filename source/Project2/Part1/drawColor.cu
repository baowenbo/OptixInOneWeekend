#include "hip/hip_runtime.h"
#include <optix.h>
#include <optixu/optixu_math_namespace.h>

using namespace optix;

rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );
rtBuffer<float3, 2> resultBuffer;

rtDeclareVariable(float3, color, , );

RT_PROGRAM void drawColor()
{
    resultBuffer[launch_index] = color;
}

