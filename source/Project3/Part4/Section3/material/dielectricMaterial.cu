#include "hip/hip_runtime.h"
#include <optix.h>
#include <optixu/optixu_math_namespace.h>

#include "core/raydata.cuh"
#include "random/sampling.cuh"

// Ray state variables
rtDeclareVariable(optix::Ray, theRay, rtCurrentRay, );
rtDeclareVariable(PerRayData, thePrd, rtPayload,  );

// "Global" variables
rtDeclareVariable(rtObject, sysWorld, , );

// The point and normal of intersection
rtDeclareVariable(HitRecord, hitRecord, attribute hitRecord, );

// Material variables
rtDeclareVariable(float, eta, , );

inline __device__ float schlickDielectric(
    const float costheta, const float eta_1, const float eta_2)
{
    float r0 = (eta_1-eta_2) / (eta_1+eta_2);
    r0 = r0*r0;
    return r0 + (1-r0)*pow((1-costheta), 5);
}

inline __device__ bool refract(const float3& v, const float3& n, 
    const float eta_1, const float eta_2, float3& refracted)
{
    float3 uv = optix::normalize(v);
    float dt = optix::dot(uv, n);
    float discriminant = 1.0f; - (eta_1/eta_2)/(eta_1/eta_2)*(1-dt*dt);

    if (discriminant > 0.0f)
    {
        refracted = (eta_1/eta_2)*(uv - dt*n) - sqrt(discriminant)*n;
        return true;
    }
    return false;
}

RT_PROGRAM void closestHit()
{
    // Determine if inside or outside of object
    float3 localNormal, unitDirection;
    float costheta, eta_1, eta_2;
    if (optix::dot(theRay.direction, hitRecord.normal) < 0.0f)
    {
        // Outside the object
        localNormal = hitRecord.normal;
        unitDirection = optix::normalize(theRay.direction);
        eta_1 = 1.0f;
        eta_2 = eta;
        costheta = optix::dot(-unitDirection, hitRecord.normal);
    }
    else
    {
        // Inside the object
        localNormal = -hitRecord.normal;
        unitDirection = optix::normalize(theRay.direction);
        eta_1 = eta;
        eta_2 = 1.0f;
        costheta = optix::dot(unitDirection, hitRecord.normal);
        costheta = sqrt(1.0f - eta_1*eta_1*(1.0f - costheta*costheta));
    }
    
    float3 refracted;
    float reflectProb;
    if (refract(theRay.direction, localNormal, eta_1, eta_2, refracted))
        reflectProb = schlickDielectric(costheta, eta_1, eta_2);
    else
        reflectProb = 1.0f;

    float3 scatterDirection;
    if(randf(thePrd.seed) < reflectProb)
        scatterDirection = optix::reflect(unitDirection, localNormal);
    else
        scatterDirection = refracted; 

    thePrd.scatterEvent = Ray_Hit;
    thePrd.scatter = optix::make_Ray(
        hitRecord.point,
        scatterDirection,
        theRay.ray_type,
        theRay.tmin,
        theRay.tmax
    );
    thePrd.attenuation = make_float3(1.0f, 1.0f, 1.0f);
}

