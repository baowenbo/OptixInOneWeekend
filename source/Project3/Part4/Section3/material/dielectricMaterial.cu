#include "hip/hip_runtime.h"
#include <optix.h>
#include <optixu/optixu_math_namespace.h>

#include "core/raydata.cuh"
#include "random/sampling.cuh"

// Ray state variables
rtDeclareVariable(optix::Ray, theRay, rtCurrentRay, );
rtDeclareVariable(PerRayData, thePrd, rtPayload,  );

// "Global" variables
rtDeclareVariable(rtObject, sysWorld, , );

// The point and normal of intersection
rtDeclareVariable(HitRecord, hitRecord, attribute hitRecord, );

// Material variables
rtDeclareVariable(float, eta, , );

inline __device__ float schlick(const float costheta, const float eta_2)
{
    float eta_1 = 1.0f;  // Assuming eta_1 is a vacuum
    float r0 = (eta_1-eta_2) / (eta_1+eta_2);
    r0 = r0*r0;
    return r0 + (1-r0)*pow((1-costheta), 5);
}

RT_PROGRAM void closestHit()
{
    float3 uv = optix::normalize(theRay.direction);
    float costheta = optix::dot(uv, hitRecord.normal);
    float3 localNormal;
    float dt, eta_iOverEta_t;
    // Inside the object
    if (costheta > 0.0f)
    {
       localNormal = -hitRecord.normal;
       dt = -costheta;
       eta_iOverEta_t = eta;
       costheta = eta * costheta;
    }
    // Outside the object
    else
    {
        localNormal = hitRecord.normal;
        dt = costheta;
        eta_iOverEta_t = 1.0f / eta; // assuming eta_i is a vacuum
        costheta = -costheta;
    }

    float discriminant = 1.0f - eta_iOverEta_t*eta_iOverEta_t*(1-dt*dt);
    float reflectProb;
    if (discriminant > 0.0f)
        reflectProb = schlick(costheta, eta);
    else
        reflectProb = 1.0f;

    float3 scatterDirection;
    if(randf(thePrd.seed) < reflectProb)
        scatterDirection = optix::reflect(uv, localNormal);
    else
        scatterDirection = eta_iOverEta_t*(uv-localNormal*dt)
            - localNormal*sqrt(discriminant);

    thePrd.scatterEvent = Ray_Hit;
    thePrd.scatter = optix::make_Ray(
        hitRecord.point,
        scatterDirection,
        theRay.ray_type,
        theRay.tmin,
        theRay.tmax
    );
    thePrd.attenuation = make_float3(1.0f, 1.0f, 1.0f);
}

