#include "hip/hip_runtime.h"
#include <optix.h>

#include "core/raydata.cuh"
#include "random/sampling.cuh"

// Ray state variables
rtDeclareVariable(optix::Ray, theRay, rtCurrentRay, );
rtDeclareVariable(PerRayData, thePrd, rtPayload,  );

// "Global" variables
rtDeclareVariable(rtObject, sysWorld, , );

// The point and normal of intersection
rtDeclareVariable(HitRecord, hitRecord, attribute hitRecord, );

// Material variables
rtDeclareVariable(float3, color, , );

RT_PROGRAM void closestHit()
{
    float3 scatterDirection = hitRecord.normal + randomInUnitSphere(thePrd.seed); 

    thePrd.scatterEvent = Ray_Hit;
    thePrd.scatter = optix::make_Ray(
        hitRecord.point,
        scatterDirection,
        theRay.ray_type,
        theRay.tmin,
        theRay.tmax
    );
    thePrd.attenuation = color;
}

