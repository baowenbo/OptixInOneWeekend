
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>

// Kernel function to determine depth of mandelbrot at cr, ci
__device__ unsigned int mandelDepth(float cr, float ci, int maxDepth)
{
    float zr = 0.0f;
    float zi = 0.0f;
    float zrSqr = 0.0f;
    float ziSqr = 0.0f;

    unsigned int i;

    for (i = 0; i < maxDepth; i++)
    {
        zi = zr * zi;
        zi += zi + ci;
        zr = zrSqr - ziSqr + cr;
        zrSqr = zr * zr;
        ziSqr = zi * zi;

        if (zrSqr + ziSqr > 4.0f) break;
    }

    return i;
}

__device__ float3 color(unsigned int depth, unsigned int maxDepth)
{
    if(depth == maxDepth)
        return make_float3(0.0f, 0.0f, 0.0f);
    else
        return make_float3(1.0f, 1.0f, 1.0f);
}

__global__ void mandel(float* buffer, 
    float xMin, float xMax,
    float yMin, float yMax,
    unsigned int maxDepth)
{
    int nx = blockDim.x;
    int ny = gridDim.x;
    float dx = (xMax - xMin) / nx;
    float dy = (yMax - yMin) / ny;
    float x = xMin + (threadIdx.x + 0.5f) * dx;
    float y = yMin + (blockIdx.x + 0.5f) * dy;

    unsigned int depth = mandelDepth(x, y, maxDepth);
    float3 depthColor = color(depth, maxDepth);

    int index = blockIdx.x*blockDim.x + threadIdx.x;
    buffer[3*index + 0] = depthColor.x;
    buffer[3*index + 1] = depthColor.y;
    buffer[3*index + 2] = depthColor.z;
}


int main(void)
{
    float xMin = -2.0f;
    float xMax = 1.0f;
    float yMin = -1.0f;
    float yMax = 1.0f;
    float step = 0.005f; 

    int nx = (xMax - xMin) / step;
    int ny = (yMax - yMin) / step;
    float* buffer;

    // Allocate Unified Memory – accessible from CPU or GPU
    hipMallocManaged(&buffer, 3*nx*ny*sizeof(float));

    // Run kernel on nx*ny elements on the GPU
    mandel<<<ny, nx>>>(buffer, 
        xMin, xMax,
        yMin, yMax,
        100
    );

    // Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();

    // Print the buffer out to command
    std::cout << "P3\n" << nx << " " << ny << "\n255\n";
    for (int j = ny-1; j >= 0; j--)
        for (int i = 0; i < nx; i++)
        {
            float* triplet = buffer + 3*(j*nx + i);
            int ir = int(255.99f * triplet[0]);
            int ig = int(255.99f * triplet[1]);
            int ib = int(255.99f * triplet[2]);
            std::cout << ir << " " << ig << " " << ib << "\n";
        }

    // Free memory
    hipFree(buffer);
  
    return 0;
}
