
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>

// Kernel function to color the buffer according to the gradient
__global__ void insideCircle(bool *buffer)
{
    int nx = blockDim.x;
    int ny = gridDim.x;

    float r = 1.0f;
    float dx = 2.0f / nx;
    float dy = 2.0f / ny;

    float x = (threadIdx.x - nx/2.0f + 0.5f) * dx;
    float y = (blockIdx.x - ny/2.0f + 0.5f) * dy;

    buffer[blockIdx.x*blockDim.x + threadIdx.x] = (x*x + y*y) < r*r;
}

int main(void)
{
    int nx = 1024;
    int ny = 1024;
    bool *buffer;

    // Allocate Unified Memory – accessible from CPU or GPU
    hipMallocManaged(&buffer, nx*ny*sizeof(bool));

    // Run kernel on nx*ny elements on the GPU
    insideCircle<<<ny, nx>>>(buffer);

    // Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();

    // Print the buffer out to command
    float sum = 0.0f;
    std::cout << "P3\n" << nx << " " << ny << "\n255\n";
    for (int j = ny-1; j >= 0; j--)
        for (int i = 0; i < nx; i++)
        {
            int index = j*nx + i;
            if(buffer[index])
            {
                std::cout << "255 255 0\n";
                sum++;
            }
            else
                std::cout << "0 0 255\n";
        }

    std::cout << "Pi is approximately " << sum * (4.0f/(nx*ny)) << std::endl;

    // Free memory
    hipFree(buffer);
  
    return 0;
}
